#include "hip/hip_runtime.h"
#include <Renderer/PathTracer.cuh>
#include <Core/KernelHelper.cuh>
#include <Core/Tracing.cuh>
#include <Geometry/Sphere.cuh>
#include <Geometry/Plane.cuh>

namespace detail
{
    __global__ void
    pathtracer_kernel(const Scene scene,
                      const uint32_t scene_size,
                      const Camera camera,
                      const uint32_t frameIndex,
                      const uint32_t maxTraceDepth,
                      Image<Vector3float> img)
    {
        const uint32_t tid = ThreadHelper::globalThreadIndex();

        if(tid >= img.size())
        {
            return;
        }

        uint32_t seed = Math::tea<4>(tid, frameIndex);

        Ray ray = Tracing::launchRay(tid, img.width(), img.height(), camera);
        
        //Scene
        Vector3float lightPos(0.0f,0.9f,2.0f);

        uint32_t trace_depth = 0;
        Vector3float radiance = 0;
        Vector3float brdf = 0;
        Vector3float rayweight = 1;
        bool continueTracing;

        do
        {
            continueTracing = false;

            //Direct illumination
            LocalGeometry geom = Tracing::traceRay(scene, scene_size, ray);
            if(geom.depth == INFINITY)break;
            Vector3float normal = geom.N;

            Vector3float inc_dir = Math::normalize(ray.origin() - geom.P);
            Vector3float lightDir = Math::normalize(lightPos - geom.P);
            float d = Math::norm(lightPos - geom.P);
            Vector3float lightRadiance = Vector3float(10.0f) / (d*d);

            Ray shadow_ray = Ray(geom.P + 0.01f*lightDir, lightDir);

            if(Tracing::traceVisibility(scene, scene_size, d, shadow_ray))
            {
                radiance += fmaxf(0.0f, Math::dot(normal,lightDir))*geom.material.brdf(geom.P,inc_dir,lightDir,normal)*lightRadiance*rayweight;
            }

            //Indirect illumination
            switch(geom.material.type)
            {
                case PHONG:
                case LAMBERT:
                {
                    const float xi_1 = Math::rnd(seed);
                    const float xi_2 = Math::rnd(seed);

                    const float r = sqrtf(xi_1);
                    const float phi = 2.0f*3.14159f*xi_2;

                    const float x = r*cos(phi);
                    const float y = r*sin(phi);
                    const float z = sqrtf(fmaxf(0.0f, 1.0f - x*x-y*y));

                    Vector3float direction = Math::normalize(Math::toLocalFrame(normal, Vector3float(x,y,z)));
                    ray = Ray(geom.P + 0.01f*direction, direction);

                    float p = fmaxf(EPSILON, Math::dot(direction, normal))/3.14159f;
                    rayweight = rayweight * fmaxf(EPSILON, Math::dot(direction, normal))*geom.material.brdf(geom.P, inc_dir, direction, normal)/p;

                    continueTracing = true;
                }
                break;
                /*case PHONG:
                {

                }
                break;*/
                case MIRROR:
                {

                }
                break;
                case GLASS:
                {

                }
                break;
            }

            ++trace_depth;
        }while(trace_depth < maxTraceDepth && continueTracing);

        img[tid] = radiance;
    }
}

void
PBRendering::pathtracing(const Scene scene,
                         const uint32_t& scene_size,
                         const Camera& camera,
                         const uint32_t& frameIndex,
                         const uint32_t& maxTraceDepth,
                         Image<Vector3float>* output_img)
{
    const KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(output_img->size());
    detail::pathtracer_kernel<<<config.blocks, config.threads>>>(scene, 
                                                                 scene_size, 
                                                                 camera,
                                                                 frameIndex,
                                                                 maxTraceDepth, 
                                                                 *output_img);
    cudaSafeCall(hipDeviceSynchronize());
}
