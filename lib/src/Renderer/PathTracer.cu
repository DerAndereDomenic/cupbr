#include "hip/hip_runtime.h"
#include <Renderer/PathTracer.cuh>
#include <Core/KernelHelper.cuh>
#include <Core/Tracing.cuh>
#include <Geometry/Sphere.cuh>
#include <Geometry/Plane.cuh>

namespace detail
{
    __global__ void
    pathtracer_kernel(const Scene scene,
                      const Camera camera,
                      const uint32_t frameIndex,
                      const uint32_t maxTraceDepth,
                      Image<Vector3float> img)
    {
        const uint32_t tid = ThreadHelper::globalThreadIndex();

        if(tid >= img.size())
        {
            return;
        }

        uint32_t seed = Math::tea<4>(tid, frameIndex);

        Ray ray = Tracing::launchRay(tid, img.width(), img.height(), camera, true, &seed);

        uint32_t trace_depth = 0;
        Vector3float radiance = 0;
        Vector3float rayweight = 1;
        Vector3float inc_dir, lightDir, lightRadiance;
        float d;

        Light light;

        do
        {
            //Direct illumination
            LocalGeometry geom = Tracing::traceRay(scene, ray);
            if(geom.depth == INFINITY)break;
            Vector3float normal = geom.N;

            inc_dir = Math::normalize(ray.origin() - geom.P);

            uint32_t light_sample = static_cast<uint32_t>(Math::rnd(seed) * scene.light_count);
            light = *(scene.lights[light_sample]); 

            switch(light.type)
            {
                case LightType::POINT:
                {
                    lightDir = Math::normalize(light.position - geom.P);
                    d = Math::norm(light.position - geom.P);
                    lightRadiance = light.intensity / (d*d);
                }
                break;
                case LightType::AREA:
                {
                    float xi1 = Math::rnd(seed) * 2.0f - 1.0f;
                    float xi2 = Math::rnd(seed) * 2.0f - 1.0f;

                    Vector3float sample = light.position + xi1 * light.halfExtend1 + xi2 * light.halfExtend2;
                    Vector3float n = Math::normalize(Math::cross(light.halfExtend1, light.halfExtend2));
                    float area = 4.0f*Math::norm(light.halfExtend1) * Math::norm(light.halfExtend2);

                    lightDir = Math::normalize(sample - geom.P);
                    d = Math::norm(sample - geom.P);

                    float NdotL = Math::dot(lightDir, n);
                    if(NdotL < 0) NdotL *= -1.0f;

                    float solidAngle =  area * NdotL / (d*d);

                    lightRadiance = light.radiance * solidAngle;
                }
                break;
            }
            

            Ray shadow_ray = Ray(geom.P + 0.01f*lightDir, lightDir);

            if(Tracing::traceVisibility(scene, d, shadow_ray))
            {
                radiance += scene.light_count*fmaxf(0.0f, Math::dot(normal,lightDir))*geom.material.brdf(geom.P,inc_dir,lightDir,normal)*lightRadiance*rayweight;
            }

            //Indirect illumination
            Vector4float direction_p = geom.material.sampleDirection(seed, inc_dir, geom.N);
            Vector3float direction = Vector3float(direction_p);
            //TODO: Remove special treatment for glass
            if(geom.material.type != GLASS)
                rayweight = rayweight * fmaxf(EPSILON, Math::dot(direction, normal))*geom.material.brdf(geom.P, inc_dir, direction, normal)/direction_p.w;
                 
            ray = Ray(geom.P+0.01f*direction, direction);

            ++trace_depth;
        }while(trace_depth < maxTraceDepth);

        if(frameIndex > 0)
        {
            const float a = 1.0f/(static_cast<float>(frameIndex) + 1.0f);
            radiance = (1.0f-a)*img[tid] + a*radiance;
        }

        img[tid] = radiance;
    }
}

void
PBRendering::pathtracing(const Scene scene,
                         const Camera& camera,
                         const uint32_t& frameIndex,
                         const uint32_t& maxTraceDepth,
                         Image<Vector3float>* output_img)
{
    const KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(output_img->size());
    detail::pathtracer_kernel<<<config.blocks, config.threads>>>(scene, 
                                                                 camera,
                                                                 frameIndex,
                                                                 maxTraceDepth, 
                                                                 *output_img);
    cudaSafeCall(hipDeviceSynchronize());
}
