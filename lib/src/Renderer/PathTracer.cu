#include "hip/hip_runtime.h"
#include <Renderer/PathTracer.cuh>
#include <Core/KernelHelper.cuh>
#include <Core/Tracing.cuh>
#include <Geometry/Sphere.cuh>
#include <Geometry/Plane.cuh>

namespace detail
{
    __global__ void
    pathtracer_kernel(const Scene scene,
                      const uint32_t scene_size,
                      const Camera camera,
                      const uint32_t maxTraceDepth,
                      Image<Vector3float> img)
    {
        const uint32_t tid = ThreadHelper::globalThreadIndex();

        if(tid >= img.size())
        {
            return;
        }
    }
}

void
PBRendering::pathtracing(const Scene scene,
                         const uint32_t& scene_size,
                         const Camera& camera,
                         const uint32_t& maxTraceDepth,
                         Image<Vector3float>* output_img)
{
    const KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(output_img->size());
    detail::pathtracer_kernel<<<config.blocks, config.threads>>>(scene, 
                                                                 scene_size, 
                                                                 camera, 
                                                                 maxTraceDepth, 
                                                                 *output_img);
    cudaSafeCall(hipDeviceSynchronize());
}
