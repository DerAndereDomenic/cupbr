#include "hip/hip_runtime.h"
#include <Renderer/PathTracer.cuh>
#include <Core/KernelHelper.cuh>
#include <Core/Tracing.cuh>
#include <Geometry/Sphere.cuh>
#include <Geometry/Plane.cuh>

namespace detail
{
    __global__ void
    pathtracer_kernel(const Scene scene,
                      const Camera camera,
                      const uint32_t frameIndex,
                      const uint32_t maxTraceDepth,
                      Image<Vector3float> img)
    {
        const uint32_t tid = ThreadHelper::globalThreadIndex();

        if(tid >= img.size())
        {
            return;
        }

        uint32_t seed = Math::tea<4>(tid, frameIndex);

        Ray ray = Tracing::launchRay(tid, img.width(), img.height(), camera);

        uint32_t trace_depth = 0;
        Vector3float radiance = 0;
        Vector3float rayweight = 1;
        Vector3float direction = 0;
        Vector3float inc_dir, lightDir, lightRadiance;
        bool continueTracing;
        float p;
        float d;

        Light light;

        do
        {
            continueTracing = false;

            //Direct illumination
            LocalGeometry geom = Tracing::traceRay(scene, ray);
            if(geom.depth == INFINITY)break;
            Vector3float normal = geom.N;


            uint32_t light_sample = static_cast<uint32_t>(Math::rnd(seed) * scene.light_count);
            light = *(scene.lights[light_sample]); 

            switch(light.type)
            {
                case LightType::POINT:
                {
                    inc_dir = Math::normalize(ray.origin() - geom.P);
                    lightDir = Math::normalize(light.position - geom.P);
                    d = Math::norm(light.position - geom.P);
                    lightRadiance = light.intensity / (d*d);
                }
                break;
                case LightType::AREA:
                {

                }
                break;
            }
            

            Ray shadow_ray = Ray(geom.P + 0.01f*lightDir, lightDir);

            if(Tracing::traceVisibility(scene, d, shadow_ray))
            {
                radiance += scene.light_count*fmaxf(0.0f, Math::dot(normal,lightDir))*geom.material.brdf(geom.P,inc_dir,lightDir,normal)*lightRadiance*rayweight;
            }

            //Indirect illumination
            switch(geom.material.type)
            {
                case PHONG:
                case LAMBERT:
                {
                    const float xi_1 = Math::rnd(seed);
                    const float xi_2 = Math::rnd(seed);

                    const float r = sqrtf(xi_1);
                    const float phi = 2.0f*3.14159f*xi_2;

                    const float x = r*cos(phi);
                    const float y = r*sin(phi);
                    const float z = sqrtf(fmaxf(0.0f, 1.0f - x*x-y*y));

                    direction = Math::normalize(Math::toLocalFrame(normal, Vector3float(x,y,z)));

                    p = fmaxf(EPSILON, Math::dot(direction, normal))/3.14159f;

                    rayweight = rayweight * fmaxf(EPSILON, Math::dot(direction, normal))*geom.material.brdf(geom.P, inc_dir, direction, normal)/p;
                    continueTracing = true;
                }
                break;
                /*case PHONG:
                {

                }
                break;*/
                case MIRROR:
                {
                    direction = Math::reflect(inc_dir, normal);

                    p = 1.0f;

                    rayweight = rayweight * fmaxf(EPSILON, Math::dot(direction, normal))*geom.material.brdf(geom.P, inc_dir, direction, normal)/p;
                    continueTracing = true;
                }
                break;
                case GLASS:
                {
                    const float NdotV = Math::dot(inc_dir, geom.N);
                    bool outside = NdotV > 0.0f;
                    float eta = outside ? 1.0f/geom.material.eta : geom.material.eta;
                    Vector3float normal = outside ? geom.N : -1.0f*geom.N;
                    float F0 = outside ? (1.0f - geom.material.eta) / (1.0f + geom.material.eta) : (-1.0f + geom.material.eta) / (1.0f + geom.material.eta);
                    F0 *= F0;

                    float p_reflect = Math::fresnel_schlick(F0, Math::dot(inc_dir, normal));
                    float xi = Math::rnd(seed);

                    Vector3float refraction_dir = Math::refract(eta, inc_dir, normal);
                    if(xi <= p_reflect || Math::safeFloatEqual(Math::norm(refraction_dir), 0.0f))
                    {
                        direction = Math::reflect(inc_dir, normal);
                    }
                    else
                    {
                        rayweight = Math::dot(normal, refraction_dir)/Math::dot(normal, inc_dir) * rayweight;
                        direction = refraction_dir;
                    }
                    
                    continueTracing = true;
                }
                break;
            }
            ray = Ray(geom.P+0.01f*direction, direction);

            ++trace_depth;
        }while(trace_depth < maxTraceDepth && continueTracing);

        if(frameIndex > 0)
        {
            const float a = 1.0f/(static_cast<float>(frameIndex) + 1.0f);
            radiance = (1.0f-a)*img[tid] + a*radiance;
        }

        img[tid] = radiance;
    }
}

void
PBRendering::pathtracing(const Scene scene,
                         const Camera& camera,
                         const uint32_t& frameIndex,
                         const uint32_t& maxTraceDepth,
                         Image<Vector3float>* output_img)
{
    const KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(output_img->size());
    detail::pathtracer_kernel<<<config.blocks, config.threads>>>(scene, 
                                                                 camera,
                                                                 frameIndex,
                                                                 maxTraceDepth, 
                                                                 *output_img);
    cudaSafeCall(hipDeviceSynchronize());
}
