#include "hip/hip_runtime.h"
#include <Renderer/GradientDomain.cuh>
#include <Core/KernelHelper.cuh>
#include <Core/Tracing.cuh>
#include <Geometry/Sphere.cuh>
#include <Geometry/Plane.cuh>

namespace detail
{
    __device__ void
    traceImage( const uint32_t& tid,
                Scene& scene,
                const Camera& camera,
                const uint32_t& frameIndex,
                const uint32_t& maxTraceDepth,
                Image<Vector3float> img)
    {
        uint32_t seed = Math::tea<4>(tid, frameIndex);

        Ray ray = Tracing::launchRay(tid, img.width(), img.height(), camera, true, &seed);

        uint32_t trace_depth = 0;
        Vector3float radiance = 0;
        Vector3float rayweight = 1;
        Vector3float inc_dir, lightDir, lightRadiance;
        float d;

        Light light;

        do
        {
            //Direct illumination
            LocalGeometry geom = Tracing::traceRay(scene, ray);
            if(geom.depth == INFINITY)
            {
                if(scene.useEnvironmentMap)
                {
                    Vector2uint32_t pixel = Tracing::direction2UV(ray.direction(), scene.environment.width(), scene.environment.height());
                    radiance += rayweight * scene.environment(pixel);
                }
                break;
            }

            Vector3float normal = geom.N;

            inc_dir = Math::normalize(ray.origin() - geom.P);

            uint32_t useEnvironmentMap = scene.useEnvironmentMap ? 1 : 0;
            uint32_t light_sample = static_cast<uint32_t>(Math::rnd(seed) * (scene.light_count + useEnvironmentMap));

            if(light_sample != scene.light_count)
            {
                light = *(scene.lights[light_sample]); 

                switch(light.type)
                {
                    case LightType::POINT:
                    {
                        lightDir = Math::normalize(light.position - geom.P);
                        d = Math::norm(light.position - geom.P);
                        lightRadiance = light.intensity / (d*d);
                    }
                    break;
                    case LightType::AREA:
                    {
                        float xi1 = Math::rnd(seed) * 2.0f - 1.0f;
                        float xi2 = Math::rnd(seed) * 2.0f - 1.0f;

                        Vector3float sample = light.position + xi1 * light.halfExtend1 + xi2 * light.halfExtend2;
                        Vector3float n = Math::normalize(Math::cross(light.halfExtend1, light.halfExtend2));
                        float area = 4.0f*Math::norm(light.halfExtend1) * Math::norm(light.halfExtend2);

                        lightDir = Math::normalize(sample - geom.P);
                        d = Math::norm(sample - geom.P);

                        float NdotL = Math::dot(lightDir, n);
                        if(NdotL < 0) NdotL *= -1.0f;

                        float solidAngle =  area * NdotL / (d*d);

                        lightRadiance = light.radiance * solidAngle;
                    }
                    break;
                }
            }
            else // Use environment map
            {
                Vector4float sample = geom.material.sampleDirection(seed, inc_dir, geom.N);
                lightDir = Vector3float(sample);
                d = INFINITY; //TODO: Better way to do this
                Vector2uint32_t pixel = Tracing::direction2UV(lightDir, scene.environment.width(), scene.environment.height());
                lightRadiance = scene.environment(pixel)/sample.w;
            }
                
            Ray shadow_ray = Ray(geom.P + 0.01f*lightDir, lightDir);

            if(Tracing::traceVisibility(scene, d, shadow_ray))
            {
                radiance += (scene.light_count+useEnvironmentMap)*fmaxf(0.0f, Math::dot(normal,lightDir))*geom.material.brdf(geom.P,inc_dir,lightDir,normal)*lightRadiance*rayweight;
            }

            //Indirect illumination
            Vector4float direction_p = geom.material.sampleDirection(seed, inc_dir, geom.N);
            Vector3float direction = Vector3float(direction_p);
            rayweight = rayweight * fabs(Math::dot(direction, normal))*geom.material.brdf(geom.P, inc_dir, direction, normal)/direction_p.w;
                 
            ray = Ray(geom.P+0.01f*direction, direction);
            ++trace_depth;
        }while(trace_depth < maxTraceDepth);

        if(frameIndex > 0)
        {
            const float a = 1.0f/(static_cast<float>(frameIndex) + 1.0f);
            radiance = (1.0f-a)*img[tid] + a*radiance;
        }

        img[tid] = radiance;
    }

    __global__ void
    gradientdomain_kernel(Scene scene,
                          const Camera camera,
                          const uint32_t frameIndex,
                          const uint32_t maxTraceDepth,
                          Image<Vector3float> img)
    {
        const uint32_t tid = ThreadHelper::globalThreadIndex();

        if(tid >= img.size())
        {
            return;
        }

        traceImage(tid,
                   scene,
                   camera,
                   frameIndex,
                   maxTraceDepth,
                   img);
    }
}

void
PBRendering::gradientdomain(Scene& scene,
                            const Camera& camera,
                            const uint32_t& frameIndex,
                            const uint32_t& maxTraceDepth,
                            Image<Vector3float>* output_img)
{
    const KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(output_img->size());
    detail::gradientdomain_kernel<<<config.blocks, config.threads>>>(scene, 
                                                                 camera,
                                                                 frameIndex,
                                                                 maxTraceDepth, 
                                                                 *output_img);
    cudaSafeCall(hipDeviceSynchronize());
}
