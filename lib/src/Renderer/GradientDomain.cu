#include "hip/hip_runtime.h"
#include <Renderer/GradientDomain.cuh>
#include <Core/KernelHelper.cuh>
#include <Core/Tracing.cuh>
#include <Geometry/Sphere.cuh>
#include <Geometry/Plane.cuh>

namespace cupbr
{
    namespace detail
    {
#define DIFFUSE_THRESHOLD 0.1f

        struct PathData
        {
            Vector3float position;
            Vector3float normal;
            bool diffuse;

            float pdf;
            bool valid;
        };

        struct RadiancePayload
        {
            uint32_t seed;
            Vector3float radiance = 0;
            Vector3float rayweight = 1;
            Vector3float out_dir;
            bool next_ray_valid;
            uint32_t trace_depth = 0;

            PathData path[10]; //Hard code max trace depth
            PathData* base_path;
            bool shift = false; //If this is a shift path
        };

        inline __device__
            void emissiveIllumintationGDPT(Ray& ray, LocalGeometry& geom)
        {
            RadiancePayload* payload = ray.payload<RadiancePayload>();

            payload->radiance += payload->rayweight * geom.material.albedo_e;
        }

        inline __device__
            void directIlluminationGDPT(Scene& scene, Ray& ray, LocalGeometry& geom, Vector3float& inc_dir)
        {
            //Direct illumination
            RadiancePayload* payload = ray.payload<RadiancePayload>();

            Vector3float normal = geom.N;

            //Don't shade back facing geometry
            if (geom.material.type != GLASS && Math::dot(normal, inc_dir) <= 0.0f)
            {
                payload->rayweight = 0;
                return;
            }

            uint32_t useEnvironmentMap = scene.useEnvironmentMap ? 1 : 0;
            uint32_t light_sample = static_cast<uint32_t>(Math::rnd(payload->seed) * (scene.light_count + useEnvironmentMap));

            Light light;
            Vector3float lightDir, lightRadiance;
            float d;
            if (light_sample != scene.light_count)
            {
                light = *(scene.lights[light_sample]);

                switch (light.type)
                {
                case LightType::POINT:
                {
                    lightDir = Math::normalize(light.position - geom.P);
                    d = Math::norm(light.position - geom.P);
                    lightRadiance = light.intensity / (d * d);
                }
                break;
                case LightType::AREA:
                {
                    float xi1 = Math::rnd(payload->seed) * 2.0f - 1.0f;
                    float xi2 = Math::rnd(payload->seed) * 2.0f - 1.0f;

                    Vector3float sample = light.position + xi1 * light.halfExtend1 + xi2 * light.halfExtend2;
                    Vector3float n = Math::normalize(Math::cross(light.halfExtend1, light.halfExtend2));
                    float area = 4.0f * Math::norm(light.halfExtend1) * Math::norm(light.halfExtend2);

                    lightDir = Math::normalize(sample - geom.P);
                    d = Math::norm(sample - geom.P);

                    float NdotL = Math::dot(lightDir, n);
                    if (NdotL < 0) NdotL *= -1.0f;

                    float solidAngle = area * NdotL / (d * d);

                    lightRadiance = light.radiance * solidAngle;
                }
                break;
                }
            }
            else // Use environment map
            {
                Vector4float sample = geom.material.sampleDirection(payload->seed, inc_dir, geom.N);
                lightDir = Vector3float(sample);
                d = INFINITY; //TODO: Better way to do this
                Vector2uint32_t pixel = Tracing::direction2UV(lightDir, scene.environment.width(), scene.environment.height());
                lightRadiance = scene.environment(pixel) / sample.w;
            }

            Ray shadow_ray = Ray(geom.P + 0.001f * lightDir, lightDir);

            if (Tracing::traceVisibility(scene, d, shadow_ray))
            {
                payload->radiance += (scene.light_count + useEnvironmentMap) *
                    fmaxf(0.0f, Math::dot(normal, lightDir)) *
                    geom.material.brdf(geom.P, inc_dir, lightDir, normal) *
                    lightRadiance *
                    payload->rayweight;
            }
        }

        inline __device__
            void indirectIlluminationGDPT(Ray& ray, LocalGeometry& geom, Vector3float& inc_dir)
        {
            //Indirect illumination
            RadiancePayload* payload = ray.payload<RadiancePayload>();
            Vector4float direction_p = geom.material.sampleDirection(payload->seed, inc_dir, geom.N);
            Vector3float direction = Vector3float(direction_p);
            if (Math::norm(direction) == 0)
                return;
            ray.payload<RadiancePayload>()->rayweight = ray.payload<RadiancePayload>()->rayweight *
                fabs(Math::dot(direction, geom.N)) *
                geom.material.brdf(geom.P, inc_dir, direction, geom.N) / direction_p.w;
            payload->out_dir = direction;
            payload->next_ray_valid = true;
            payload->path[payload->trace_depth].pdf = direction_p.w;
        }

        __device__ void
            collect_radiance(Ray& ray,
                Scene& scene,
                const Camera& camera,
                const uint32_t& maxTraceDepth)
        {
            RadiancePayload* payload = ray.payload<RadiancePayload>();

            uint32_t trace_depth = 0;
            Vector3float inc_dir;

            Light light;

            do
            {
                payload->next_ray_valid = false;

                //Return if no valid base vertex is found for reversibility
                if (payload->shift && !payload->base_path[payload->trace_depth].valid) return;
                LocalGeometry geom = Tracing::traceRay(scene, ray);
                if (geom.depth == INFINITY)
                {
                    payload->path[payload->trace_depth].valid = false;
                    if (scene.useEnvironmentMap)
                    {
                        Vector2uint32_t pixel = Tracing::direction2UV(ray.direction(), scene.environment.width(), scene.environment.height());
                        payload->radiance += payload->rayweight * scene.environment(pixel);
                    }
                    break;
                }

                //Store path data
                payload->path[payload->trace_depth].position = geom.P;
                payload->path[payload->trace_depth].normal = geom.N;
                payload->path[payload->trace_depth].valid = true;
                payload->path[payload->trace_depth].diffuse = geom.material.shininess > DIFFUSE_THRESHOLD;

                Vector3float inc_dir = -1.0f * ray.direction();

                emissiveIllumintationGDPT(ray, geom);
                directIlluminationGDPT(scene, ray, geom, inc_dir);
                indirectIlluminationGDPT(ray, geom, inc_dir);

                ray.traceNew(geom.P + 0.01f * payload->out_dir, payload->out_dir);

                if (!payload->next_ray_valid)break;
                ++trace_depth;
                ++payload->trace_depth;
            } while (trace_depth < maxTraceDepth);
        }

        __global__ void
            gdpt_kernel(Scene scene,
                Camera camera,
                const uint32_t frameIndex,
                const uint32_t maxTraceDepth,
                Image<Vector3float> img,
                Image<Vector3float> gX_forward,
                Image<Vector3float> gX_backward,
                Image<Vector3float> gY_forward,
                Image<Vector3float> gY_backward)
        {
            const uint32_t tid = ThreadHelper::globalThreadIndex();

            if (tid >= img.size())
            {
                return;
            }

            const Vector2int32_t pixel = ThreadHelper::index2pixel(tid, img.width(), img.height());
            if (pixel.x == 0 || pixel.x == img.width() - 1 || pixel.y == 0 || pixel.y == img.height() - 1)
            {
                return;
            }

            uint32_t seed = Math::tea<4>(tid, frameIndex);

            Ray base_ray = Tracing::launchRay(pixel, img.width(), img.height(), camera, true, &seed);
            RadiancePayload payload_base;
            payload_base.seed = seed;
            base_ray.setPayload(&payload_base);

            Ray left_ray = Tracing::launchRay(pixel + Vector2int32_t(-1, 0), img.width(), img.height(), camera, true, &seed);
            RadiancePayload payload_left;
            payload_left.seed = seed;
            payload_left.base_path = payload_base.path;
            payload_left.shift = true;
            left_ray.setPayload(&payload_left);

            Ray right_ray = Tracing::launchRay(pixel + Vector2int32_t(1, 0), img.width(), img.height(), camera, true, &seed);
            RadiancePayload payload_right;
            payload_right.seed = seed;
            payload_right.base_path = payload_base.path;
            payload_right.shift = true;
            right_ray.setPayload(&payload_right);

            Ray up_ray = Tracing::launchRay(pixel + Vector2int32_t(0, 1), img.width(), img.height(), camera, true, &seed);
            RadiancePayload payload_up;
            payload_up.seed = seed;
            payload_up.base_path = payload_base.path;
            payload_up.shift = true;
            up_ray.setPayload(&payload_up);

            Ray down_ray = Tracing::launchRay(pixel + Vector2int32_t(0, -1), img.width(), img.height(), camera, true, &seed);
            RadiancePayload payload_down;
            payload_down.seed = seed;
            payload_down.base_path = payload_base.path;
            payload_down.shift = true;
            down_ray.setPayload(&payload_down);

            collect_radiance(base_ray, scene, camera, maxTraceDepth);
            collect_radiance(left_ray, scene, camera, maxTraceDepth);
            collect_radiance(right_ray, scene, camera, maxTraceDepth);
            collect_radiance(up_ray, scene, camera, maxTraceDepth);
            collect_radiance(down_ray, scene, camera, maxTraceDepth);

            Vector3float radiance = payload_base.radiance;
            Vector3float gradient_x_forward = 0.5f * (radiance - payload_left.radiance);
            Vector3float gradient_x_backward = 0.5f * (payload_right.radiance - radiance);
            Vector3float gradient_y_forward = 0.5f * (radiance - payload_down.radiance);
            Vector3float gradient_y_backward = 0.5f * (payload_up.radiance - radiance);

            if (frameIndex > 0)
            {
                const float a = 1.0f / (static_cast<float>(frameIndex) + 1.0f);
                radiance = (1.0f - a) * img[tid] + a * radiance;
                gradient_x_forward = (1.0f - a) * gX_forward[tid] + a * gradient_x_forward;
                gradient_x_backward = (1.0f - a) * gX_backward[tid] + a * gradient_x_backward;
                gradient_y_forward = (1.0f - a) * gY_forward[tid] + a * gradient_y_forward;
                gradient_y_backward = (1.0f - a) * gY_backward[tid] + a * gradient_y_backward;
            }

            img[tid] = radiance;
            gX_forward[tid] = gradient_x_forward;
            gX_backward[tid] = gradient_x_backward;
            gY_forward[tid] = gradient_y_forward;
            gY_backward[tid] = gradient_y_backward;
        }

        __global__ void
            init_kernel(Image<Vector3float> reconstruction,
                Image<Vector3float> base,
                Image<Vector3float> gradient_x_forward,
                Image<Vector3float> gradient_x_backward,
                Image<Vector3float> gradient_y_forward,
                Image<Vector3float> gradient_y_backward,
                Image<Vector3float> gradient_x,
                Image<Vector3float> gradient_y)
        {
            const uint32_t tid = ThreadHelper::globalThreadIndex();

            if (tid >= reconstruction.size())
            {
                return;
            }

            reconstruction[tid] = base[tid];

            const Vector2uint32_t pixel = ThreadHelper::index2pixel(tid, reconstruction.width(), reconstruction.height());

            if (pixel.x == 0 || pixel.y == 0 || pixel.x == reconstruction.width() - 1 || pixel.y == reconstruction.height() - 1)return;

            gradient_x[tid] = gradient_x_forward[tid] + gradient_x_backward[tid];
            gradient_y[tid] = gradient_y_forward[tid] + gradient_y_backward[tid];
        }

        inline __device__ float
            compute_median(float* values)
        {
            float median = 0.0f;
            uint32_t arg_min = 0;
            uint32_t arg_max = 0;

#define INNER_LOOP(i)\
            median += values[i];\
            if(values[i] < values[arg_min])\
            {\
                arg_min = i;\
            }\
            if (values[i] > values[arg_max])\
            {\
                arg_max = i;\
            }

            INNER_LOOP(0);
            INNER_LOOP(1);
            INNER_LOOP(2);
            INNER_LOOP(3);

#undef INNER_LOOP

            return (median - values[arg_min] - values[arg_max]) / 2.0f;
        }

        __global__ void
            optimization_kernel(Image<Vector3float> reconstruction,
                Image<Vector3float> gradient_x,
                Image<Vector3float> gradient_y,
                Image<Vector3float> temp)
        {
            const uint32_t tid = ThreadHelper::globalThreadIndex();

            if (tid >= reconstruction.size())
            {
                return;
            }

            const Vector2uint32_t pixel = ThreadHelper::index2pixel(tid, reconstruction.width(), reconstruction.height());

            if (pixel.x == 0 || pixel.y == 0 || pixel.x == reconstruction.width() - 1 || pixel.y == reconstruction.height() - 1)return;

            Vector2uint32_t pixel_left(pixel.x - 1, pixel.y);
            Vector2uint32_t pixel_right(pixel.x + 1, pixel.y);
            Vector2uint32_t pixel_up(pixel.x, pixel.y + 1);
            Vector2uint32_t pixel_down(pixel.x, pixel.y - 1);

            Vector3float v1 = reconstruction(pixel_left) + gradient_x(pixel_left);
            Vector3float v2 = reconstruction(pixel_down) + gradient_y(pixel_down);
            Vector3float v3 = reconstruction(pixel_right) - gradient_x[tid];
            Vector3float v4 = reconstruction(pixel_up) - gradient_y[tid];

            float values_red[4] =
            {
                v1.x, v2.x, v3.x, v4.x
            };

            float values_green[4] =
            {
                v1.y, v2.y, v3.y, v4.y
            };

            float values_blue[4] =
            {
                v1.z, v2.z, v3.z, v4.z
            };

            float med_red = compute_median(values_red);
            float med_green = compute_median(values_green);
            float med_blue = compute_median(values_blue);

            temp[tid] = Vector3float(med_red, med_green, med_blue);
        }
    } //namespace detail

    void
        PBRendering::gradientdomain(Scene& scene,
            const Camera& camera,
            const uint32_t& frameIndex,
            const uint32_t& maxTraceDepth,
            Image<Vector3float>* base,
            Image<Vector3float>* temp,
            Image<Vector3float>* gradient_x,
            Image<Vector3float>* gradient_y,
            Image<Vector3float>* gradient_x_forward,
            Image<Vector3float>* gradient_x_backward,
            Image<Vector3float>* gradient_y_forward,
            Image<Vector3float>* gradient_y_backward,
            Image<Vector3float>* output_img)
    {
        const KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(output_img->size());
        detail::gdpt_kernel << <config.blocks, config.threads >> > (scene,
            camera,
            frameIndex,
            maxTraceDepth,
            *base,
            *gradient_x_forward,
            *gradient_x_backward,
            *gradient_y_forward,
            *gradient_y_backward);
        cudaSafeCall(hipDeviceSynchronize());

        detail::init_kernel << <config.blocks, config.threads >> > (*output_img,
            *base,
            *gradient_x_forward,
            *gradient_x_backward,
            *gradient_y_forward,
            *gradient_y_backward,
            *gradient_x,
            *gradient_y);

        cudaSafeCall(hipDeviceSynchronize());

        //Optimization
        for (uint32_t i = 0; i < 50; ++i)
        {
            detail::optimization_kernel << <config.blocks, config.threads >> > (*output_img,
                *gradient_x,
                *gradient_y,
                *temp);
            cudaSafeCall(hipDeviceSynchronize());

            temp->copyDevice2DeviceObject(*output_img);
        }
    }

} //namespace cupbr
