#include "hip/hip_runtime.h"
#include <Scene/SceneLoader.cuh>
#include <Core/Memory.cuh>
#include <Geometry/Plane.cuh>
#include <Geometry/Sphere.cuh>

Scene
SceneLoader::cornellBoxSphere()
{
    Scene scene;
    scene.scene_size = 8;
    scene.light_count = 1;
    scene.geometry = Memory::allocator()->createDeviceArray<Geometry*>(scene.scene_size);
    scene.lights = Memory::allocator()->createDeviceArray<Light*>(scene.light_count);

    Plane h_floor(Vector3float(0,-1,0), Vector3float(0,1,0));
    Plane h_ceil(Vector3float(0,1,0), Vector3float(0,-1,0));
    Plane h_left(Vector3float(-1,0,0), Vector3float(1,0,0));
    Plane h_right(Vector3float(1,0,0), Vector3float(-1,0,0));
    Plane h_back(Vector3float(0,0,3), Vector3float(0,0,-1)); 

    Sphere h_diff(Vector3float(-0.65f,-0.75f,2.65f), 0.25f);
    Sphere h_mirror(Vector3float(0.65f,-0.75f,2.65f), 0.25f);
    Sphere h_glass(Vector3float(0.0f,-0.75f,1.25f), 0.25f);

    Light h_light;
    h_light.position = Vector3float(0.0f, 0.9f, 2.0f);
    h_light.intensity = 1;

    Plane* floor = Memory::allocator()->createDeviceObject<Plane>();
    Plane* ceil = Memory::allocator()->createDeviceObject<Plane>();
    Plane* left = Memory::allocator()->createDeviceObject<Plane>();
    Plane* right = Memory::allocator()->createDeviceObject<Plane>();
    Plane* back = Memory::allocator()->createDeviceObject<Plane>();

    Sphere* diff = Memory::allocator()->createDeviceObject<Sphere>();
    Sphere* mirror = Memory::allocator()->createDeviceObject<Sphere>();
    Sphere* glass = Memory::allocator()->createDeviceObject<Sphere>();

    Light* light = Memory::allocator()->createDeviceObject<Light>();

    h_left.material.albedo_d = Vector3float(0,1,0);
    h_right.material.albedo_d = Vector3float(1,0,0);
    h_diff.material.albedo_d = Vector3float(0,0,1);
    h_diff.material.albedo_s = Vector3float(1,1,1);
    h_diff.material.type = MaterialType::PHONG;
    h_mirror.material.type = MaterialType::MIRROR;
    h_mirror.material.albedo_s = 1;
    h_glass.material.type = MaterialType::GLASS;
    h_glass.material.albedo_s = 1;

    Memory::allocator()->copyHost2DeviceObject<Plane>(&h_floor, floor);
    Memory::allocator()->copyHost2DeviceObject<Plane>(&h_ceil, ceil);
    Memory::allocator()->copyHost2DeviceObject<Plane>(&h_left, left);
    Memory::allocator()->copyHost2DeviceObject<Plane>(&h_right, right);
    Memory::allocator()->copyHost2DeviceObject<Plane>(&h_back, back);

    Memory::allocator()->copyHost2DeviceObject<Sphere>(&h_diff, diff);
    Memory::allocator()->copyHost2DeviceObject<Sphere>(&h_mirror, mirror);
    Memory::allocator()->copyHost2DeviceObject<Sphere>(&h_glass, glass);

    Memory::allocator()->copyHost2DeviceObject<Light>(&h_light, light);

    Geometry* host_array[] = {floor, ceil, left, right, back, diff, mirror, glass};
    Light* host_lights[] = {light};

    Memory::allocator()->copyHost2DeviceArray<Geometry*>(scene.scene_size, host_array, scene.geometry);
    Memory::allocator()->copyHost2DeviceArray<Light*>(scene.light_count, host_lights, scene.lights);

    return scene;
}

void
SceneLoader::destroyCornellBoxSphere(Scene scene)
{
    Geometry* host_scene[8];
    Light* host_lights[1];
    Memory::allocator()->copyDevice2HostArray(8, scene.geometry, host_scene);
    Memory::allocator()->copyDevice2HostArray(1, scene.lights, host_lights);

    for(uint32_t i = 0; i < 5; ++i)
    {
        Memory::allocator()->destroyDeviceObject<Plane>(static_cast<Plane*>(host_scene[i]));
    }

    for(uint32_t i = 5; i < 8; ++i)
    {
        Memory::allocator()->destroyDeviceObject<Sphere>(static_cast<Sphere*>(host_scene[i]));
    }

    for(uint32_t i = 0; i < 1; ++i)
    {
        Memory::allocator()->destroyDeviceObject<Light>(host_lights[i]);
    }

    Memory::allocator()->destroyDeviceArray<Geometry*>(scene.geometry);
    Memory::allocator()->destroyDeviceArray<Light*>(scene.lights);
}