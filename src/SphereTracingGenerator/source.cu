#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include <CUPBR.h>
#include <CUNET.h>

#include <Models/LenGen.h>

using namespace cupbr;

/*struct MediumSettings
{
    float sigma;
    float phi;
    float g;
};

struct PathSummary
{
    uint32_t N;
    Vector3float x;
    Vector3float w;
    Vector3float X;
    Vector3float W;
};

struct FileLine
{
    float sigma;
    float g;
    float phi;
    uint32_t N;
    float cosTheta;
    float beta;
    float alpha;
    float Xx;
    float Xy;
    float Xz;
    float Wx;
    float Wy;
    float Wz;
};

__device__ MediumSettings generateNewSettings(uint32_t& seed)
{
    float densityRnd = Math::rnd(seed);
    float scatterAlbedoRnd = Math::rnd(seed);
    float gRnd = Math::rnd(seed);

    float density = (densityRnd * densityRnd * densityRnd) * 300.0f;
    float p = scatterAlbedoRnd * scatterAlbedoRnd * scatterAlbedoRnd;
    float scatterAlbedo = fminf(1.0f, 1.000001f - p * p);
    float g = Math::clamp(gRnd * 2.0f - 1.0f, -0.999f, 0.999f);

    return {density, scatterAlbedo, g};
}

__device__ float invertcdf(const float& g, const float& xi)
{
    float t = (1.0f - g * g) / (1.0f - g + 2.0f * g * xi);
    return 0.5f * (1 + g * g - t * t) / g;
}

__device__ void createOrthoBasis(const Vector3float& N, Vector3float& T, Vector3float& B)
{
    float sign = N.z / fabsf(N.z);
    float a = -1.0f / (sign + N.z);
    float b = N.x * N.y * a;
    T = Vector3float(1.0f + sign * N.x * N.x * a, sign * b, -sign * N.x);
    B = Vector3float(b, sign + N.y * N.y * a, -N.y);
}

__device__ float distanceToBoundary(const Vector3float& x, const Vector3float& d)
{
    float b = 2.0f * Math::dot(x,d);
    float c = Math::dot(x,x) - 1.0f;

    float disc = b * b - 4 * c;
    if (disc <= 0)
        return 0;

    return fmaxf(0.0f, (-b + sqrtf(disc)) / 2.0f);
}

__device__ Vector3float samplePhase(const Vector3float& w, const float& g, uint32_t& seed)
{
    if(abs(g) < 0.001f)
    {
        //Random direction
        float r1 = Math::rnd(seed);
        float r2 = Math::rnd(seed) * 2.0f - 1.0f;
        float sqrR2 = r2 * r2;
        float two_pi_by_r1 = 2.0f * M_PI * r1;
        float sqrt_of_one_minus_sqrR2 = sqrtf(1.0f - sqrR2);
        float x = cosf(two_pi_by_r1) * sqrt_of_one_minus_sqrR2;
        float y = sinf(two_pi_by_r1) * sqrt_of_one_minus_sqrR2;
        float z = r2;

        Vector3float t0, t1;
        createOrthoBasis(-1.0f * w, t0, t1);
        return t0 * x + t1 * y + (-1.0f * w)*z;
    }

    float phi = Math::rnd(seed) * 2 * M_PI;
    float cosTheta = invertcdf(g, Math::rnd(seed));
    float sinTheta = sqrtf(fmaxf(0.0f, 1.0f - cosTheta * cosTheta));

    Vector3float t0, t1;
    createOrthoBasis(w, t0, t1);

    return sinTheta * sinf(phi) * t0 + sinTheta * cosf(phi) * t1 + cosTheta * w;
}

__device__ PathSummary getVPTSampleInSphere(MediumSettings& settings, uint32_t& seed)
{
    Vector3float x = 0;
    Vector3float w(0,0,1);
    Vector3float X = x;
    Vector3float W = w;
    uint32_t N = 0;
    float accum = 0;
    float importance = 1;

    while(true)
    {
        importance *= settings.phi;
        accum += importance;

        if(Math::rnd(seed) < importance / accum)
        {
            X = x;
            W = w;
        }

        w = samplePhase(w, settings.g, seed);
        ++N;

        float d = distanceToBoundary(x,w);

        float t = settings.sigma < 0.00001f ? 10000000.0f : -log(fmaxf(0.000000001f, 1.0f - Math::rnd(seed))) / settings.sigma;

        if(t>= d || isnan(t) || isinf(t))
        {
            x += w * d;
            return {N, x, w, X, W};
        }
        x += w * t;
    }
}

__global__ void generateSamples(const uint32_t N, FileLine* output)
{
    const uint32_t tid = ThreadHelper::globalThreadIndex();

    if(tid >= N)
    {
        return;
    }

    uint32_t seed = Math::tea<4>(tid, 0);

    MediumSettings settings = generateNewSettings(seed);
    PathSummary r = getVPTSampleInSphere(settings, seed);

    Vector3float zAxis = Vector3float(0,0,1);
    Vector3float xAxis = fabsf(r.x.z) > 0.999 ? Vector3float(1,0,0) : Math::normalize(Math::cross(r.x, Vector3float(0,0,1)));
    Vector3float yAxis = Math::cross(zAxis, xAxis);

    Matrix3x3float R = Math::transpose(Matrix3x3float(xAxis, yAxis, zAxis));

    Vector3float normx = R * r.x;
    Vector3float normw = R * r.w;
    Vector3float normX = R * r.X;
    Vector3float normW = R * r.W;
    
    Vector3float B(1,0,0);
    Vector3float T = Math::cross(normx, B);
    float cosTheta = normx.z;
    float beta = Math::dot(normw, T);
    float alpha = Math::dot(normw, B);

    output[tid] = 
    {
        settings.sigma,
        settings.g,
        settings.phi,
        r.N,
        cosTheta,
        beta,
        alpha,
        normX.x,
        normX.y,
        normX.z,
        normW.x,
        normW.y,
        normW.z
    };
}

void generateDatasetForTrainingCVAE()
{
    const uint32_t N = 1 << 22;
    printf("Generatirng %i samples\n", N);

    FileLine* dev_output = Memory::createDeviceArray<FileLine>(N);

    printf("Start generating samples...\n");
    KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(N);
    generateSamples<<<config.blocks, config.threads>>>(N, dev_output);
    cudaSafeCall(hipDeviceSynchronize());
    printf("Start writing file...\n");

    FileLine* output = Memory::createHostArray<FileLine>(N);
    Memory::copyDevice2HostArray<FileLine>(N, dev_output, output);

    std::ofstream file;
    file.open("ScattersDataSet.ds");

    for(uint32_t i = 0; i < N; ++i)
    {
        file << output[i].sigma << ", " <<
                output[i].g << ", " <<
                output[i].phi << ", " <<
                output[i].N << ", " <<
                output[i].cosTheta << ", " <<
                output[i].beta << ", " <<
                output[i].alpha << ", " <<
                output[i].Xx << ", " <<
                output[i].Xy << ", " <<
                output[i].Xz << ", " <<
                output[i].Wx << ", " << 
                output[i].Wy << ", " <<
                output[i].Wz << "\n";
    }

    file.close();

    Memory::destroyHostArray<FileLine>(output);
    Memory::destroyDeviceObject<FileLine>(dev_output);
}

__global__ void test(const uint32_t N, cunet::LenGen lenGen, float* buffer)
{
    const uint32_t tid = ThreadHelper::globalThreadIndex();

    if (tid >= N)
        return;

    uint32_t seed = Math::tea<4>(tid, 0);

    Vector2float lenLatent = Math::sampleStdNormal2D(seed);
    float lenInput[4] = {10, 0.9, lenLatent.x, lenLatent.y};
    float lenOutput[2];
    lenGen(lenInput, lenOutput);

    float logN = fmaxf(0.0f, lenOutput[0] + Math::sampleStdNormal1D(seed) * expf(0.5*Math::clamp(lenOutput[1], -16.0f, 16.0f)));
    float n = roundf(expf(logN) + 0.49f);

    buffer[tid] = n;
}

int run()
{
    cudaSafeCall(hipSetDevice(0));

    uint32_t N = 1 << 22;

    cunet::LenGen lenGen_host;
    cunet::LenGen* lenGen = cunet::Memory::createDeviceObject<cunet::LenGen>();

    cunet::Memory::copyHost2DeviceObject<cunet::LenGen>(&lenGen_host, lenGen);

    float* buffer = Memory::createDeviceArray<float>(N);

    KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(N);
    test << <config.blocks, config.threads >> > (N, lenGen_host, buffer);
    hipDeviceSynchronize();

    float* host_buffer = Memory::createHostArray<float>(N);
    Memory::copyDevice2HostArray<float>(N, buffer, host_buffer);
    Memory::destroyDeviceArray<float>(buffer);

    std::ofstream file;
    file.open("Evaluation.ds");

    for(uint32_t i = 0; i < N; ++i)
    {
        file << host_buffer[i] << "\n";
    }

    file.close();


    Memory::destroyHostArray<float>(host_buffer);
    //generateDatasetForTrainingCVAE();

    return 0;
}

int main()
{
    int exit = run();
    return exit;
}*/

struct PathSummary
{
    Vector3float inc_pos;
    Vector3float inc_dir;
    Vector3float out_pos;
    Vector3float out_dir;
    uint32_t num_scattering;
};

struct DataPoint
{
    float theta_dir_in;
    uint32_t num_scattering;
    float theta_pos_out;
    float phi_pos_out;
    float theta_dir_out;
    float phi_dir_out;
};

__device__ float distanceToBoundary(const Vector3float& x, const Vector3float& d)
{
    float b = 2.0f * Math::dot(x,d);
    float c = Math::dot(x,x) - 1.0f;

    float disc = b * b - 4 * c;
    if (disc <= 0)
        return 0;

    return fmaxf(0.0f, (-b + sqrtf(disc)) / 2.0f);
}

__device__ Vector3float sampleHenyeyGreensteinPhase(const float& g, const Vector3float& forward, uint32_t& seed)
{
    float u1 = Math::rnd(seed);
    float u2 = Math::rnd(seed);

    float g2 = g * g;
    float d = (1.0f - g2) / (1.0f - g + 2.0f * g * u1);
    float cos_theta = Math::clamp(0.5f / g * (1.0f + g2 - d * d), -1.0f, 1.0f);

    float sin_theta = sqrtf(fmaxf(0.0f, 1.0f - cos_theta * cos_theta));
    float phi = 2.0f * 3.14159f * u2;

    float x = sin_theta * cosf(phi);
    float y = sin_theta * sinf(phi);
    float z = cos_theta;

    Vector3float result = Math::normalize(Math::toLocalFrame(forward, Vector3float(x, y, z)));

    return result;
}

__device__ Vector3float sampleHemisphereUniform(uint32_t& seed, const Vector3float& N)
{
    float z = Math::rnd(seed) * 2.0f - 1.0f;
    float phi = Math::rnd(seed) * 2.0f * M_PI;

    float r = sqrtf(fmaxf(0.0f, 1.0f - z * z));
    float x = r * cosf(phi);
    float y = r * sinf(phi);

    Vector3float result(x, y, z);

    return Math::dot(result, N) < 0 ? -1.0f * result : result;
}

__device__ Vector3float sampleSphereUniform(uint32_t& seed)
{
    Vector3float result = sampleHemisphereUniform(seed, Vector3float(1, 0, 0));
    return Math::rnd(seed) < 0.5 ? result : -1.0f * result;
}

__global__ void generateSamples(const uint32_t num_samples, Sphere sphere, PathSummary* buffer)
{
    const uint32_t tid = ThreadHelper::globalThreadIndex();

    if(tid >= num_samples)
    {
        return;
    }

    uint32_t seed = Math::tea<4>(tid, 0);

    Vector3float start_pos = Vector3float(0, 0, 0.99);
    float theta = Math::rnd(seed)* static_cast<float>(M_PI) / 2.0f;
    Vector3float direction = -1.0f * Vector3float(sinf(theta), 0, cosf(theta));
    Ray ray(start_pos, direction);
    PathSummary summary = { start_pos, direction, Vector3float(0), Vector3float(0), 0 };

    //Volume
    float sigma_s = 3.0f;
    float sigma_a = 0.0f;
    float sigma_t = sigma_s + sigma_a;
    float g = 0.6f;

    while(true)
    {
        ray.traceNew(start_pos + 0.001f * direction, direction);

        float d = distanceToBoundary(ray.origin(), ray.direction());

        if (d == INFINITY)
        {
            summary.out_pos = ray.origin();
            summary.out_dir = ray.direction();
            break;
        }

        float t = -logf(1.0f - Math::rnd(seed)) / sigma_t;

        if (t >= d)
        {
            //printf("sample\n");
            summary.out_pos = ray.origin() + d * ray.direction();
            summary.out_dir = direction;
            break;
        }

        ++summary.num_scattering;
        start_pos = ray.origin() + t * ray.direction();
        direction = sampleHenyeyGreensteinPhase(g, ray.direction(), seed);
        //path[summary.num_scattering] = start_pos;
    }

    //for(int i = 0; i < summary.num_scattering + 1; ++i)
    //{
    //    printf("%f %f %f\n", path[i].x, path[i].y, path[i].z);
    //}

    buffer[tid] = summary;
}

__global__ void convertDataSet(const uint32_t N, PathSummary* buffer, DataPoint* dataset)
{
    const uint32_t tid = ThreadHelper::globalThreadIndex();

    if(tid >= N)
    {
        return;
    }

    PathSummary summary = buffer[tid];

    float theta_dir_in = acosf(-Math::clamp(summary.inc_dir.z, -1.0f, 1.0f));

    float theta_pos_out = acosf(Math::clamp(summary.out_pos.z, -1.0f, 1.0f));
    float phi_pos_out = atan2f(summary.out_pos.y, summary.out_pos.x);

    float theta_dir_out = acosf(Math::clamp(summary.out_dir.z, -1.0f, 1.0f));
    float phi_dir_out = atan2f(summary.out_dir.y, summary.out_dir.x);
    
    dataset[tid] = {theta_dir_in, summary.num_scattering, theta_pos_out, phi_pos_out, theta_dir_out, phi_dir_out};
}

void generateDataSet()
{
    const uint32_t N = 1 << 22;
    printf("Generating %i samples...\n", N);

    Sphere sphere(Vector3float(0, 0, 0), 1);
    
    PathSummary* buffer = Memory::createDeviceArray<PathSummary>(N);
    DataPoint* dataset = Memory::createDeviceArray<DataPoint>(N);

    KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(N);
    generateSamples << <config.blocks, config.threads >> > (N, sphere, buffer);
    ::cudaSafeCall(hipDeviceSynchronize());

    convertDataSet << <config.blocks, config.threads >> > (N, buffer, dataset);
    ::cudaSafeCall(hipDeviceSynchronize());

    DataPoint* host_dataset = Memory::createHostArray<DataPoint>(N);
    Memory::copyDevice2HostArray<DataPoint>(N, dataset, host_dataset);
    Memory::destroyDeviceArray<DataPoint>(dataset);

    std::ofstream file;
    file.open("SphereScatters_Ver2.ds");

    for(uint32_t i = 0; i < N; ++i)
    {
        file << host_dataset[i].theta_dir_in << ", " <<
                host_dataset[i].num_scattering << ", " <<
                host_dataset[i].theta_pos_out << ", " <<
                host_dataset[i].phi_pos_out << ", " <<
                host_dataset[i].theta_dir_out << ", " <<
                host_dataset[i].phi_dir_out << "\n";
    }

    file.close();

    Memory::destroyHostArray<DataPoint>(host_dataset);
    Memory::destroyDeviceArray<PathSummary>(buffer);
}

int run()
{
    cudaSafeCall(hipSetDevice(0));

    generateDataSet();

    return 0;
}

int main()
{
    int exit = run();
    return exit;
}