#include "hip/hip_runtime.h"
#include <CUPBR.h>

namespace cupbr
{
    
    class SDFDifference : public SDF
    {
        public:

        SDFDifference(Properties* properties)
        {
            lhs = static_cast<SDF*>(properties->getProperty("lhs", (void*)nullptr));
            rhs = static_cast<SDF*>(properties->getProperty("rhs", (void*)nullptr));
        }

        CUPBR_HOST_DEVICE
        virtual float operator()(const Vector3float& x) 
        { 
            return fmaxf((*lhs)(x), -(*rhs)(x));
        }

        private:
        SDF* lhs;
        SDF* rhs;
    };

    DEFINE_PLUGIN(SDFDifference, "SDFDifference", "1.0", SDF)

}