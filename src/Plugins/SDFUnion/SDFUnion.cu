#include "hip/hip_runtime.h"
#include <CUPBR.h>

namespace cupbr
{
    
    class SDFUnion : public SDF
    {
        public:

        SDFUnion(Properties* properties)
        {
            lhs = static_cast<SDF*>(properties->getProperty("lhs", (void*)nullptr));
            rhs = static_cast<SDF*>(properties->getProperty("rhs", (void*)nullptr));
        }

        CUPBR_HOST_DEVICE
        virtual float operator()(const Vector3float& x) 
        { 
            return fminf((*lhs)(x), (*rhs)(x));
        }

        private:
        SDF* lhs;
        SDF* rhs;
    };

    DEFINE_PLUGIN(SDFUnion, "SDFUnion", "1.0", SDF)

}