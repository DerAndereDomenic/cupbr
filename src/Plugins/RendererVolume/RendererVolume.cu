#include "hip/hip_runtime.h"
#include <CUPBR.h>

namespace cupbr
{

    namespace detail
    {
        struct RadiancePayload
        {
            uint32_t seed;
            Vector3float radiance = 0;
            Vector3float rayweight = 1;
            Vector3float ray_start;
            Vector3float out_dir;
            bool next_ray_valid;
            Volume* volume;
            bool inside_object = false;
            uint32_t object_index = 0;
        };

        CUPBR_DEVICE void directIlluminationVolumetric(GeometryScene& scene, Ray& ray, LocalGeometry& geom, Vector3float& inc_dir)
        {
            //Direct illumination
            RadiancePayload* payload = ray.payload<RadiancePayload>();

            Vector3float normal = geom.N;

            //Don't shade back facing geometry
            if (geom.material->type != MaterialType::REFRACTIVE && Math::dot(normal, inc_dir) <= 0.0f)
            {
                //payload->rayweight = 0;
                return;
            }

            uint32_t useEnvironmentMap = scene.useEnvironmentMap ? 1 : 0;
            uint32_t light_sample = static_cast<uint32_t>(Math::rnd(payload->seed) * (scene.light_count + useEnvironmentMap));

            Light light;
            Vector3float lightDir, lightRadiance;
            float d;
            if (light_sample != scene.light_count)
            {
                light = *(scene.lights[light_sample]);

                lightRadiance = light.sample(payload->seed, geom.P, lightDir, d);
            }
            else // Use environment map
            {
                Vector4float sample = geom.material->sampleDirection(payload->seed, inc_dir, geom.N);
                lightDir = Vector3float(sample);
                d = INFINITY; //TODO: Better way to do this
                Vector2uint32_t pixel = Tracing::direction2UV(lightDir, scene.environment.width(), scene.environment.height());
                lightRadiance = scene.environment(pixel) / sample.w;
            }

            Ray shadow_ray = Ray(geom.P + 0.001f * lightDir, lightDir);

            if (Tracing::traceVisibility(scene, d, shadow_ray))
            {
                payload->radiance += (scene.light_count + useEnvironmentMap) *
                    fmaxf(0.0f, Math::dot(normal, lightDir)) *
                    Math::exp(-1.0f*(payload->volume->sigma_s + payload->volume->sigma_a) * fminf(d, 100000000.0f)) *
                    geom.material->brdf(geom.P, inc_dir, lightDir, normal) *
                    lightRadiance *
                    payload->rayweight;
            }
        }

        CUPBR_DEVICE void indirectIlluminationVolumetric(Ray& ray, LocalGeometry& geom, Vector3float& inc_dir)
        {
            RadiancePayload* payload = ray.payload<RadiancePayload>();

            //Kind of back-face culling to handle back facing shading normals
            if (geom.material->type != MaterialType::REFRACTIVE && Math::dot(geom.N, inc_dir) <= 0.0f)
            {
                payload->ray_start = geom.P;
                payload->out_dir = -1.0f * inc_dir;
                payload->next_ray_valid = true;
                return;
            }

            //Indirect illumination
            Vector4float direction_p = geom.material->sampleDirection(payload->seed, inc_dir, geom.N);
            Vector3float direction = Vector3float(direction_p);
            if (Math::norm(direction) == 0)
                return;
            ray.payload<RadiancePayload>()->rayweight = ray.payload<RadiancePayload>()->rayweight *
                fabs(Math::dot(direction, geom.N)) *
                geom.material->brdf(geom.P, inc_dir, direction, geom.N) / direction_p.w;
            payload->out_dir = direction;
            payload->ray_start = geom.P;
            payload->next_ray_valid = true;
        }

        CUPBR_DEVICE bool handleMediumInteraction(GeometryScene& scene, Ray& ray, LocalGeometry& geom, Vector3float& inc_dir)
        {
            RadiancePayload* payload = ray.payload<RadiancePayload>();

            float g = payload->volume->g;
            Vector3float sigma_a = payload->volume->sigma_a;
            Vector3float sigma_s = payload->volume->sigma_s;
            Vector3float sigma_t = sigma_a + sigma_s;

            uint32_t channel = static_cast<uint32_t>(Math::rnd(payload->seed) * 3);

            if (Math::safeFloatEqual(sigma_t[channel], 0.0f))
                return false;

            float t =  - logf(1.0f - Math::rnd(payload->seed)) / sigma_t[channel];

            if (t < geom.depth)
            {
                Vector3float event_position = ray.origin() + t * ray.direction();
                float scattering_prob = sigma_s[channel] / sigma_t[channel];

                if (Math::rnd(payload->seed) < scattering_prob)
                {
                    //Attenuate ray from its start to the medium event
                    //For monochrome sigma this is 1 but it may change for mulit channel scattering

                    float pdf = 0.0f;
                    for(uint32_t i = 0; i < 3; ++i)
                    {
                        pdf += sigma_t[i] * expf(-1.0f * sigma_t[i] * t);
                    }
                    pdf /= 3.0f;

                    payload->rayweight = payload->rayweight *
                        sigma_s / scattering_prob *
                        Math::exp(-1.0f*sigma_t * t) / pdf;
                }
                else
                {
                    payload->rayweight = 0;
                    payload->next_ray_valid = false;
                    return true;
                }

                //Direct illumination
                uint32_t useEnvironmentMap = scene.useEnvironmentMap ? 1 : 0;
                uint32_t light_sample = static_cast<uint32_t>(Math::rnd(payload->seed) * (scene.light_count + useEnvironmentMap));

                Light light;
                Vector3float lightDir, lightRadiance;
                float d;
                if (light_sample != scene.light_count)
                {
                    light = *(scene.lights[light_sample]);

                    lightRadiance = light.sample(payload->seed, event_position, lightDir, d);
                }
                else // Use environment map
                {
                    Vector4float sample = geom.material->sampleDirection(payload->seed, inc_dir, geom.N);
                    lightDir = Vector3float(sample);
                    d = INFINITY; //TODO: Better way to do this
                    Vector2uint32_t pixel = Tracing::direction2UV(lightDir, scene.environment.width(), scene.environment.height());
                    lightRadiance = scene.environment(pixel) / sample.w;
                }

                Ray shadow_ray;

                // If we are inside an object -> First move to border of current object -> then do light sampling
                //TODO: This is still not optimal for some scenarios because of numerical issues (?)
                Vector3float attenuation = 1.0f;
                if(payload->inside_object)
                {
                    shadow_ray = Ray(event_position, lightDir);
                    LocalGeometry ge = Tracing::traceRay(scene, shadow_ray, payload->object_index);
                    if (ge.depth == INFINITY) 
                    {
                        ge.depth = 0;
                        ge.P = event_position;
                    }
                    attenuation = Math::exp(-1.0f * sigma_t * ge.depth) * Material::henyeyGreensteinPhaseFunction(g, -Math::dot(inc_dir,lightDir));
                    shadow_ray.traceNew(ge.P + 0.001f * lightDir, lightDir);
                    d -= ge.depth;
                }
                else
                {
                    shadow_ray = Ray(event_position + 0.001f * lightDir, lightDir);
                }

                Vector3float scene_sigma_t = scene.volume.sigma_a + scene.volume.sigma_s;

                if (Tracing::traceVisibility(scene, d, shadow_ray))
                {
                    payload->radiance += (float)(scene.light_count + useEnvironmentMap) *
                        Math::exp(-1.0f*scene_sigma_t * fminf(d, 100000000.0f)) * attenuation *
                        lightRadiance *
                        payload->rayweight;
                    //Phase/pdf = 1
                }

                //Indirect Illumination
                Vector4float sample_hg = Material::sampleHenyeyGreensteinPhaseFunction(g, -1.0f * inc_dir, payload->seed);
                payload->out_dir = Vector3float(sample_hg);
                payload->ray_start = event_position;
                //Phase/pdf = 1

                payload->next_ray_valid = true;
                return true;
            }
            else
            {
                float pdf = 0.0f;

                for(uint32_t i = 0; i < 3; ++i)
                {
                    pdf += expf(-sigma_t[i] * geom.depth);
                }
                pdf /= 3.0f;

                payload->rayweight = payload->rayweight * Math::exp(-1.0f*sigma_t * geom.depth) / pdf;
                payload->ray_start = geom.P;
                return false;
            }
        }

        CUPBR_GLOBAL void
        volume_kernel(GeometryScene scene,
                      const Camera camera,
                      const uint32_t frameIndex,
                      const uint32_t maxTraceDepth,
                      const bool useRussianRoulette,
                      Image<Vector3float> img)
        {
            const uint32_t tid = ThreadHelper::globalThreadIndex();

            if (tid >= img.size())
            {
                return;
            }

            uint32_t seed = Math::tea<4>(tid, frameIndex);

            Ray ray = Tracing::launchRay(tid, img.width(), img.height(), camera, true, &seed);
            RadiancePayload payload;
            payload.seed = seed;
            payload.volume = &(scene.volume);
            ray.setPayload(&payload);

            uint32_t trace_depth = 0;

            Light light;

            do
            {
                payload.next_ray_valid = false;
                LocalGeometry geom;
                    
                if(!payload.inside_object)
                {
                    geom = Tracing::traceRay(scene, ray);
                }
                else
                {
                    geom = Tracing::traceRay(scene, ray, payload.object_index);
                }

                Vector3float inc_dir = -1.0f * ray.direction(); //Points away from surface
                if (geom.depth == INFINITY)
                {
                    if (scene.useEnvironmentMap)
                    {
                        Vector2uint32_t pixel = Tracing::direction2UV(ray.direction(), scene.environment.width(), scene.environment.height());
                        payload.radiance += payload.rayweight * scene.environment(pixel);
                    }
                    break;
                }

                if (!handleMediumInteraction(scene, ray, geom, inc_dir))
                {
                    //Handle medium interfaces
                    if(geom.material->type == MaterialType::REFRACTIVE)
                    {
                        payload.out_dir = geom.material->sampleDirection(payload.seed, inc_dir, geom.N);
                        payload.ray_start = geom.P;
                        bool reflect = Math::dot(inc_dir, geom.N) * Math::dot(payload.out_dir, geom.N) > 0;
                        payload.inside_object = reflect ? payload.inside_object : !payload.inside_object;
                        payload.object_index = geom.scene_index;
                        payload.next_ray_valid = true;
                        payload.volume = payload.inside_object ? &(geom.material->volume) : &(scene.volume);
                    }
                    else
                    {
                        directIlluminationVolumetric(scene, ray, geom, inc_dir);
                        indirectIlluminationVolumetric(ray, geom, inc_dir);
                    }

                    if(useRussianRoulette)
                    {
                        float alpha = Math::clamp(fmaxf(payload.rayweight.x, fmaxf(payload.rayweight.y, payload.rayweight.z)), 0.0f, 1.0f);
                        if(Math::rnd(payload.seed) > alpha || Math::safeFloatEqual(alpha, 0))
                        {
                            payload.next_ray_valid = false;
                            payload.rayweight = 0;
                            break;
                        }
                        payload.rayweight = payload.rayweight / alpha;
                    }
                    
                }
                ray.traceNew(payload.ray_start + 0.001f * payload.out_dir, payload.out_dir);
                if (!payload.next_ray_valid)break;
                ++trace_depth;
            } while (trace_depth < maxTraceDepth);

            if (frameIndex > 0)
            {
                const float a = 1.0f / (static_cast<float>(frameIndex) + 1.0f);
                ray.payload<RadiancePayload>()->radiance = (1.0f - a) * img[tid] + a * ray.payload<RadiancePayload>()->radiance;
            }

            img[tid] = ray.payload<RadiancePayload>()->radiance;
        }
    } //namespace detail
    
    class RendererVolume : public RenderMethod
    {
        public:

        RendererVolume(Properties* properties)
        {
            max_trace_depth = properties->getProperty("max_trace_depth", 5);
            use_russian_roulette = properties->getProperty("use_russian_roulette", true);
        }

        virtual void 
        render(Scene* scene,
               const Camera& camera,
               const uint32_t& frameIndex,
               Image<Vector3float>* output_img) 
        {
            GeometryScene* geom_scene = dynamic_cast<GeometryScene*>(scene);

            if (geom_scene == nullptr)
            {
                std::cerr << "ERROR: VolumeRenderer received scene that does not hold geometry information!\n";
                return;
            }

            const KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(output_img->size());
            detail::volume_kernel << <config.blocks, config.threads >> > (*geom_scene,
                                                                          camera,
                                                                          frameIndex,
                                                                          max_trace_depth,
                                                                          use_russian_roulette,
                                                                          *output_img);
            synchronizeDefaultStream();
        }
        
        private:
        uint32_t max_trace_depth;
        bool use_russian_roulette;
    };

    DEFINE_PLUGIN(RendererVolume, "VolumeRenderer", "1.0", RenderMethod)

}