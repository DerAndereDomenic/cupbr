#include <iostream>
#include <chrono>

#include <GL/GLRenderer.cuh>
#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <Core/KernelHelper.cuh>
#include <Core/Tracing.cuh>

#include <DataStructure/Camera.cuh>

#include <Scene/SceneLoader.cuh>

#include <Renderer/ToneMapper.cuh>
#include <Renderer/PBRenderer.cuh>

int main()
{
    bool edit = true;
    bool pressed = false;
    const uint32_t width = 1024, height = 1024;

    cudaSafeCall(hipSetDevice(0));

    Scene scene = SceneLoader::cornellBoxSphereMultiLight();

    PBRenderer pbrenderer(PATHTRACER);
    pbrenderer.setOutputSize(width, height);
    pbrenderer.registerScene(scene);

    KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(pbrenderer.getOutputImage()->size());
    ToneMapper reinhard_mapper(REINHARD);
    ToneMapper gamma_mapper(GAMMA);
    reinhard_mapper.registerImage(pbrenderer.getOutputImage());
    gamma_mapper.registerImage(pbrenderer.getOutputImage());

    ToneMapper* mapper = &reinhard_mapper;

    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(width, height, "Hello World", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);

    if (glewInit() != GLEW_OK)
	{
		std::cout <<"RENDERER::GLEWINIT::ERROR\n";
	}

    GLRenderer renderer(width, height);
    Camera camera;
    float time = 0.0f;
    uint32_t frame_counter = 0;

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Render here */
        glClear(GL_COLOR_BUFFER_BIT);

        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
        pbrenderer.render(camera);

        mapper->toneMap();
        renderer.renderTexture(mapper->getRenderBuffer());
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        time = std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count();
        ++frame_counter;

        if(frame_counter%30 == 0)
        {
            printf("\rRender time: %fms : %ffps", time/1000.0f, 1000000.0f/time);
            fflush(stdout);
        }

        /* Swap front and back buffers */
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();

        if(!edit)
            camera.processInput(window);

        if(glfwGetKey(window, GLFW_KEY_R) == GLFW_PRESS)
        {
            mapper = &reinhard_mapper;
        }
        if(glfwGetKey(window, GLFW_KEY_G) == GLFW_PRESS)
        {
            mapper = &gamma_mapper;
        }

        if(glfwGetKey(window, GLFW_KEY_1) == GLFW_PRESS && !pressed)
        {
            pbrenderer.setMethod(RAYTRACER);
        }

        if(glfwGetKey(window, GLFW_KEY_2) == GLFW_PRESS && !pressed)
        {
            pbrenderer.setMethod(WHITTED);
        }

        if(glfwGetKey(window, GLFW_KEY_3) == GLFW_PRESS && !pressed)
        {
            pbrenderer.setMethod(PATHTRACER);
        }

        if(glfwGetKey(window, GLFW_KEY_LEFT_ALT) == GLFW_PRESS && !pressed)
        {
            pressed = true;
            edit = !edit;
            if(edit)
            {
                glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
            }
            else
            {
                glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
            }
        }

        if(glfwGetKey(window, GLFW_KEY_LEFT_ALT) == GLFW_RELEASE)
        {
            pressed = false;
        }

        if(glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        {
            glfwDestroyWindow(window);
            break;
        }
    }
    printf("\n");

    glfwTerminate();

    SceneLoader::destroyCornellBoxSphere(scene);

    //TODO
    reinhard_mapper.~ToneMapper();
    gamma_mapper.~ToneMapper();
    pbrenderer.~PBRenderer();

    Memory::allocator()->printStatistics();

    return 0;
}