#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLRenderer.cuh>
#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <Core/KernelHelper.cuh>
#include <DataStructure/Camera.cuh>

#include <Geometry/Sphere.cuh>
#include <Geometry/Plane.cuh>

__global__ void fillBuffer(RenderBuffer img, const Camera camera)
{
    uint32_t tid = ThreadHelper::globalThreadIndex();

    if(tid >= img.size())
    {
        return;
    }

    const float width = img.width();
    const float height = img.height();
    const Vector2uint32_t pixel = ThreadHelper::index2pixel(tid, width, height);

    const float ratio_x = 2.0f*(static_cast<float>(pixel.x)/width - 0.5f);
    const float ratio_y = 2.0f*(static_cast<float>(pixel.y)/height - 0.5f);

    const Vector3float world_pos = camera.position() + camera.zAxis() + ratio_x*camera.xAxis() + ratio_y*camera.yAxis();

    Sphere sphere(Vector3float(0,0,2), 1);
    Plane plane(Vector3float(0,-1,0), Vector3float(0,1,0));
    Ray ray(camera.position(), world_pos - camera.position());
    
    //Compute intersection
    Vector4float intersection_sphere = sphere.computeRayIntersection(ray);
    Vector4float intersection_plane = plane.computeRayIntersection(ray);
    Vector4float intersection = intersection_plane.w < intersection_sphere.w ? intersection_plane : intersection_sphere;
    Vector3float intersection_point = Vector3float(intersection);
    Vector3float normal = intersection_plane.w < intersection_sphere.w ? plane.getNormal(intersection_point) : plane.getNormal(intersection_point);

    //Lighting
    const Vector3float lightPos(1,2,-2);

    Vector3float brdf = Vector3float(1,1,1)/static_cast<float>(M_PI); //Albedo/pi
    Vector3float lightIntensity = Vector3float(10,10,10); //White light
    Vector3float lightDir = Math::normalize(lightPos - intersection_point);
    float d = Math::norm(intersection_point-lightPos);
    Vector3float lightRadiance = lightIntensity/(d*d);
    float cosTerm = max(0.0f,Math::dot(normal, lightDir));
    Vector3float radiance = brdf*lightRadiance*cosTerm;

    //Shadow
    if(intersection.w != INFINITY)
    {
        Ray shadow_ray(intersection_point-EPSILON*ray.direction(), lightDir);
        Vector4float shadow_sphere = sphere.computeRayIntersection(shadow_ray);
        Vector4float shadow_plane = plane.computeRayIntersection(shadow_ray);

        if(shadow_sphere.w != INFINITY || shadow_plane.w != INFINITY)radiance = Vector3float(0);
    }

    //Tone mapping

    Vector3uint8_t color(0);

    if(intersection.w != INFINITY)
    {
        float mapped_red = powf(1.0 - expf(-radiance.x), 1.0f/2.2f);
        float mapped_green = powf(1.0 - expf(-radiance.y), 1.0f/2.2f);
        float mapped_blue = powf(1.0 - expf(-radiance.z), 1.0f/2.2);

        uint8_t red = mapped_red > 1.0 ? 255 : static_cast<uint8_t>(mapped_red*255.0f);
        uint8_t green = mapped_green > 1.0 ? 255 : static_cast<uint8_t>(mapped_green*255.0f);
        uint8_t blue = mapped_blue > 1.0 ? 255 : static_cast<uint8_t>(mapped_blue*255.0f);

        color = Vector3uint8_t(red, green, blue);
    }

    img[tid] = Vector4uint8_t(color,255);
}

int main()
{
    bool edit = true;
    bool pressed = false;
    const uint32_t width = 1024, height = 1024;

    cudaSafeCall(hipSetDevice(0));

    RenderBuffer img = RenderBuffer::createDeviceObject(width, height);
    KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(img.size());

    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(width, height, "Hello World", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);

    if (glewInit() != GLEW_OK)
	{
		std::cout <<"RENDERER::GLEWINIT::ERROR\n";
	}

    GLRenderer renderer = GLRenderer::createHostObject(width, height);
    Camera camera;

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Render here */
        glClear(GL_COLOR_BUFFER_BIT);

        fillBuffer<<<config.blocks, config.threads>>>(img,camera);
        cudaSafeCall(hipDeviceSynchronize());
        renderer.renderTexture(img);

        /* Swap front and back buffers */
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();

        if(!edit)
            camera.processInput(window);

        if(glfwGetKey(window, GLFW_KEY_LEFT_ALT) == GLFW_PRESS && !pressed)
        {
            pressed = true;
            edit = !edit;
            if(edit)
            {
                glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
            }
            else
            {
                glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
            }
        }

        if(glfwGetKey(window, GLFW_KEY_LEFT_ALT) == GLFW_RELEASE)
        {
            pressed = false;
        }

        if(glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        {
            glfwDestroyWindow(window);
            break;
        }
    }

    glfwTerminate();

    RenderBuffer::destroyDeviceObject(img);

    Memory::allocator()->printStatistics();

    return 0;
}