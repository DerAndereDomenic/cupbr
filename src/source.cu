#include <iostream>
#include <chrono>

#include <CUPBR.cuh>

using namespace cupbr;

int run(int argc, char* argv[])
{
    bool edit = true;
    bool pressed = false;
    const uint32_t width = 1024, height = 1024;

    cudaSafeCall(hipSetDevice(0));

    Scene scene;
    
    if(argc == 1)
    {
        scene = SceneLoader::loadFromFile("res/Scenes/CornellBoxSphereAreaLight.xml");
    }
    else if(argc == 2)
    {
        scene = SceneLoader::loadFromFile(argv[1]);
    }

    PBRenderer pbrenderer(RenderingMethod::PATHTRACER);
    pbrenderer.setOutputSize(width, height);
    pbrenderer.registerScene(&scene);

    PostProcessor postprocessor;
    postprocessor.registerImage(pbrenderer.getOutputImage());

    Vector3float kernel_data[9] =
    {
        Vector3float(1.0),Vector3float(2.0),Vector3float(1.0),
        Vector3float(0),Vector3float(0),Vector3float(0),
        Vector3float(-1.0),Vector3float(-2.0),Vector3float(-1.0),
    };

    Image<Vector3float> kernel = Image<Vector3float>::createDeviceObject(kernel_data, 3, 3);

    ToneMapper mapper(ToneMappingType::REINHARD);
    mapper.registerImage(postprocessor.getPostProcessBuffer());

    Window window("CUPBR", width, height);

    GLRenderer renderer(width, height);
    Camera camera(width,height);
    Interactor interactor(pbrenderer.getMethod());
    interactor.registerWindow((GLFWwindow*)window.getInternalWindow());
    interactor.registerCamera(camera);
    interactor.registerScene(&scene);

    float time = 0.0f;
    uint32_t frame_counter = 0;

    bool post_proc = true;

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose((GLFWwindow*)window.getInternalWindow()))
    {
        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

        window.imguiBegin();

        interactor.handleInteraction();

        if(post_proc != interactor.usePostProcessing())
        {
            if(interactor.usePostProcessing())
            {
                mapper.registerImage(postprocessor.getPostProcessBuffer());
            }
            else
            {
                mapper.registerImage(pbrenderer.getOutputImage());
            }
            post_proc = interactor.usePostProcessing();
        }
        

        mapper.setExposure(interactor.getExposure());

        if(interactor.getRenderingMethod() != pbrenderer.getMethod() || interactor.updated())
        {
            pbrenderer.setMethod(interactor.getRenderingMethod());
        }

        if(interactor.getToneMapping() != mapper.getType())
        {
            mapper.setType(interactor.getToneMapping());
        }
        
        /* Render here */
        glClear(GL_COLOR_BUFFER_BIT);

        pbrenderer.render(camera);

        //postprocessor.filter(kernel);
        if(interactor.usePostProcessing())
            postprocessor.bloom(interactor.getThreshold());

        mapper.toneMap();
        renderer.renderTexture(mapper.getRenderBuffer());
        
        ++frame_counter;

        if(frame_counter%30 == 0)
        {
            printf("\rRender time: %fms : %ffps", time/1000.0f, 1000000.0f/time);
            fflush(stdout);
        }

        window.imguiEnd();

        window.spinOnce();

        if(!edit)
            camera.processInput((GLFWwindow*)window.getInternalWindow(), time);

        if(glfwGetKey((GLFWwindow*)window.getInternalWindow(), GLFW_KEY_LEFT_ALT) == GLFW_PRESS && !pressed)
        {
            pressed = true;
            edit = !edit;
            if(edit)
            {
                glfwSetInputMode((GLFWwindow*)window.getInternalWindow(), GLFW_CURSOR, GLFW_CURSOR_NORMAL);
            }
            else
            {
                glfwSetInputMode((GLFWwindow*)window.getInternalWindow(), GLFW_CURSOR, GLFW_CURSOR_DISABLED);
            }
        }

        if(glfwGetKey((GLFWwindow*)window.getInternalWindow(), GLFW_KEY_LEFT_ALT) == GLFW_RELEASE)
        {
            pressed = false;
        }

        if(glfwGetKey((GLFWwindow*)window.getInternalWindow(), GLFW_KEY_ESCAPE) == GLFW_PRESS)
        {
            glfwDestroyWindow((GLFWwindow*)window.getInternalWindow());
            break;
        }

        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        time = static_cast<float>(std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count());
    }
    printf("\n");

    SceneLoader::destroyScene(scene);

    mapper.saveToFile("bin/output.bmp");

    Image<Vector3float>::destroyDeviceObject(kernel);

    printf("Rendered Frames: %i\n", frame_counter);
    return 0;
}

int main(int argc, char* argv[])
{
    int exit =  run(argc, argv);
    Memory::printStatistics();
    return exit;
}