#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLRenderer.cuh>
#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <Core/KernelHelper.cuh>
#include <DataStructure/Camera.cuh>

#include <Geometry/Sphere.cuh>

__global__ void fillBuffer(RenderBuffer img, const Camera camera)
{
    uint32_t tid = ThreadHelper::globalThreadIndex();

    if(tid >= img.size())
    {
        return;
    }

    const float width = img.width();
    const float height = img.height();
    const Vector2uint32_t pixel = ThreadHelper::index2pixel(tid, width, height);

    const float ratio_x = 2.0f*(static_cast<float>(pixel.x)/width - 0.5f);
    const float ratio_y = 2.0f*(static_cast<float>(pixel.y)/height - 0.5f);

    const Vector3float world_pos = camera.position() + camera.zAxis() + ratio_x*camera.xAxis() + ratio_y*camera.yAxis();

    Sphere sphere(Vector3float(0,0,2), 1);
    Ray ray(camera.position(), world_pos - camera.position());
    
    //Compute intersection
    Vector4float intersection = sphere.computeRayIntersection(ray);

    //"Tone mapping"

    int8_t ratio = intersection.w == INFINITY ? 0 : 255;

    img[tid] = Vector4uint32_t(ratio, ratio, ratio,255);
}

int main()
{
    cudaSafeCall(hipSetDevice(0));

    RenderBuffer img = RenderBuffer::createDeviceObject(640, 480);
    KernelSizeHelper::KernelSize config = KernelSizeHelper::configure(img.size());

    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(640, 480, "Hello World", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);

    if (glewInit() != GLEW_OK)
	{
		std::cout <<"RENDERER::GLEWINIT::ERROR\n";
	}

    GLRenderer renderer = GLRenderer::createHostObject(640, 480);
    Camera camera;

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Render here */
        glClear(GL_COLOR_BUFFER_BIT);

        fillBuffer<<<config.blocks, config.threads>>>(img,camera);
        cudaSafeCall(hipDeviceSynchronize());
        renderer.renderTexture(img);

        /* Swap front and back buffers */
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();
    }

    glfwTerminate();

    RenderBuffer::destroyDeviceObject(img);

    Memory::allocator()->printStatistics();

    return 0;
}